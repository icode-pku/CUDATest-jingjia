#include "hip/hip_runtime.h"
#include "cudaInc.h"
#include <iostream>
#include "CBenchMark_CUDAStreamGroupImpl.h"
#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20
namespace TestCUDA
{
	__global__ void kernelAdd(int *a, int *b, int *c)
	{
		int threadID = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadID < N)
		{
			c[threadID] = (a[threadID] + b[threadID]) / 2;
		}
	}

	bool CBenchMark_CUDAStreamGroupImpl::SetupTest() 
	{
		try
		{
			(CTestBase *)(this)->Log("hipStreamCreate in...");

			hipStream_t stream;
			CudaSafeCallEx(hipStreamCreate(&stream), CBenchMark_CUDAStreamGroupImpl);

			(CTestBase *)(this)->Log("hipStreamCreate create success...");

			int *host_a, *host_b, *host_c;
			int *dev_a, *dev_b, *dev_c;

			CudaSafeCallEx(hipMalloc((void **)&dev_a, -1 * sizeof(int)), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipMalloc((void **)&dev_b, N * sizeof(int)), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipMalloc((void **)&dev_c, N * sizeof(int)), CBenchMark_CUDAStreamGroupImpl);

			CudaSafeCallEx(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CBenchMark_CUDAStreamGroupImpl);
			if(host_a == nullptr || host_b == nullptr || host_c == nullptr){
				throw CBenchMark_CUDAStreamGroupImpl("hipHostAlloc", "CPU 内存开辟失败!");
			}
			for (int i = 0; i < FULL_DATA_SIZE; i++)
			{
				host_a[i] = i;
				host_b[i] = FULL_DATA_SIZE - i;
			}

			for (int i = 0; i < FULL_DATA_SIZE; i += N)
			{
				CudaSafeCallEx(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream), CBenchMark_CUDAStreamGroupImpl);
				CudaSafeCallEx(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream), CBenchMark_CUDAStreamGroupImpl);

				kernelAdd<<<N / 1024, 1024, 0, stream>>>(dev_a, dev_b, dev_c);

				CudaSafeCallEx(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream), CBenchMark_CUDAStreamGroupImpl);
			}

			CudaSafeCallEx(hipStreamSynchronize(stream), CBenchMark_CUDAStreamGroupImpl);
			(CTestBase *)(this)->Log("hipStreamCreate excute success...");

			CudaSafeCallEx(hipHostFree(host_a), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipHostFree(host_b), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipHostFree(host_c), CBenchMark_CUDAStreamGroupImpl);

			CudaSafeCallEx(hipFree(dev_a), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipFree(dev_b), CBenchMark_CUDAStreamGroupImpl);
			CudaSafeCallEx(hipFree(dev_c), CBenchMark_CUDAStreamGroupImpl);

			CudaSafeCallEx(hipStreamDestroy(stream), CBenchMark_CUDAStreamGroupImpl);
			(CTestBase *)(this)->Log("hipStreamCreate out...");
			this->PrintLog();
		}
		catch (CBenchMark_CUDAStreamGroupImpl &err)
		{
			(CTestBase *)(this)->Log(err.what());
			this->AddError(err.what(), this->m_pip_flags);
		}
		catch(...){
			this->AddError("CUDA_STREAM_GROUP exception!!", this->m_pip_flags);
		}
		//CudaKernelCheck(CBenchMark_CUDAStreamGroupImpl);
		return true;
	}

	bool CBenchMark_CUDAStreamGroupImpl::SetupBenchMark()
	{

		return true;
	}
	bool CBenchMark_CUDAStreamGroupImpl::SetName(const char *_api_name)
	{
		this->m_api_name = std::string(_api_name);
		return true;
	}
	const char *CBenchMark_CUDAStreamGroupImpl::what(){
		return m_error.c_str();
	}
}