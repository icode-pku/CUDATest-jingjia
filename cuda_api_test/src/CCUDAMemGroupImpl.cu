#include "hip/hip_runtime.h"

#include "cudaInc.h"
#include <iostream>
#include "CCUDAMemGroupImpl.h"
#include "CTestCUDAFactory.h"
#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20
namespace TestCUDA
{
    REGISTER_TEST_OBJECT(CCUDAMemGroupImpl, CUDA_MEM_GROUP);
    __global__ void kernelAdd_mem(int *a, int *b, int *c)
    {
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < N)
        {
            c[threadID] = (a[threadID] + b[threadID]) / 2;
        }
    }
    void CCUDAMemGroupImpl::Test_normal()
    {
#ifdef _DEBUG
        LOG_DEBUG("CCUDAMemGroupImpl in...");
#endif
        hipEvent_t start_event, end_event;
        CudaIsSafeCall(hipEventCreate(&start_event), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipEventCreate(&end_event), CCUDAMemGroupImpl);
#ifdef _DEBUG
        LOG_DEBUG("CCUDAMemGroupImpl create success...");
#endif
        int *host_a, *host_b, *host_c;
        int *dev_a, *dev_b, *dev_c;

        CudaIsSafeCall(hipMalloc((void **)&dev_a, -1 * sizeof(int)), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipMalloc((void **)&dev_b, N * sizeof(int)), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipMalloc((void **)&dev_c, N * sizeof(int)), CCUDAMemGroupImpl);

        CudaIsSafeCall(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipHostMalloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAMemGroupImpl);
        if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
        {
            throw CCUDAMemGroupImpl("hipHostAlloc", "CPU 内存开辟失败!");
        }
        for (int i = 0; i < FULL_DATA_SIZE; i++)
        {
            host_a[i] = i;
            host_b[i] = FULL_DATA_SIZE - i;
        }

        CudaIsSafeCall(hipEventRecord(start_event), CCUDAMemGroupImpl);
        for (int i = 0; i < FULL_DATA_SIZE; i += N)
        {
            CudaIsSafeCall(hipMemcpy(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice), CCUDAMemGroupImpl);
            CudaIsSafeCall(hipMemcpy(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice), CCUDAMemGroupImpl);

            kernelAdd_mem<<<N / 1024, 1024>>>(dev_a, dev_b, dev_c);

            CudaIsSafeCall(hipMemcpy(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost), CCUDAMemGroupImpl);
        }
        CudaIsSafeCall(hipEventSynchronize(start_event), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipEventRecord(end_event), CCUDAMemGroupImpl);

        float time_gpu;
        CudaIsSafeCall(hipEventElapsedTime(&time_gpu, start_event, end_event), CCUDAMemGroupImpl);

        // 事件计时
        // printf("GPU time: %.3f ms \n", time_gpu);
#ifdef _DEBUG
        LOG_DEBUG("CCUDAMemGroupImpl excute success...");
#endif
        CudaIsSafeCall(hipHostFree(host_a), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipHostFree(host_b), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipHostFree(host_c), CCUDAMemGroupImpl);

        CudaIsSafeCall(hipFree(dev_a), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipFree(dev_b), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipFree(dev_c), CCUDAMemGroupImpl);

        CudaIsSafeCall(hipEventDestroy(start_event), CCUDAMemGroupImpl);
        CudaIsSafeCall(hipEventDestroy(end_event), CCUDAMemGroupImpl);
#ifdef _DEBUG
        LOG_DEBUG("CCUDAMemGroupImpl out...");
#endif
    }
    void CCUDAMemGroupImpl::Test_NonNormal()
    {
    }
}
