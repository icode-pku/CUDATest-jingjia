#include "CCUDADeviceGroupImpl.h"
#include "cudaInc.h"
#include "TestDefine.h"
#include "Logger.h"
#include "CTestCUDAFactory.h"
namespace TestCUDA
{
    REGISTER_TEST_OBJECT(CCUDADeviceGroupImpl, CUDA_DEVICE_GROUP);
    void CCUDADeviceGroupImpl::Test_normal()
    {
        int device = 0;
        CudaIsSafeCall(hipGetDevice(&device), CCUDADeviceGroupImpl);
#ifdef _DEBUG
        LOG_DEBUG("hipGetDevice ... ");
#endif
        int device_num = 0;
        CudaIsSafeCall(hipGetDeviceCount(&device_num), CCUDADeviceGroupImpl);
#ifdef _DEBUG
        LOG_DEBUG("hipGetDeviceCount ... ");
        LOG_DEBUG("api hipGetDeviceProperties ...");
#endif
        for (size_t i = 0; i < device_num; i++)
        {
            hipDeviceProp_t prop;
            CudaIsSafeCall(hipGetDeviceProperties(&prop, i), CCUDADeviceGroupImpl);
            LOG_INFO("    -- device name :%s\n", prop.name);
            LOG_INFO("    -- device global memory size :%lf Gb\n", double(prop.totalGlobalMem) / 1024.0f / 1024.0f / 1024.0f);
            LOG_INFO("    -- device sharedMemPerBlock size: %lf Kb\n", double(prop.sharedMemPerBlock) / 1024.0f);
            LOG_INFO("    -- device warpSize: %d\n", prop.warpSize);
            LOG_INFO("    -- device maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
        }
    }
    void CCUDADeviceGroupImpl::Test_NonNormal()
    {
    }
}