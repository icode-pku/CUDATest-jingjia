#include "hip/hip_runtime.h"

#include "cudaInc.h"
#include <iostream>
#include "CCUDAEventGroupImpl.h"
#include "CTestCUDAFactory.h"
#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20
namespace TestCUDA
{
    REGISTER_TEST_OBJECT(CCUDAEventGroupImpl, CUDA_EVENT_GROUP);
    __global__ void kernelAdd_event(int *a, int *b, int *c)
    {
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < N)
        {
            c[threadID] = (a[threadID] + b[threadID]) / 2;
        }
    }

    bool CCUDAEventGroupImpl::SetupTest()
    {
        BEGIN_EXCEPTION
        (this)->Log("CCUDAEventGroupImpl in...");

        hipEvent_t start_event, end_event;
        CudaSafeCallEx(hipEventCreate(&start_event), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipEventCreate(&end_event), CCUDAEventGroupImpl);

        (CTestBase *)(this)->Log("CCUDAEventGroupImpl create success...");

        int *host_a, *host_b, *host_c;
        int *dev_a, *dev_b, *dev_c;

        CudaSafeCallEx(hipMalloc((void **)&dev_a, N * sizeof(int)), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipMalloc((void **)&dev_b, N * sizeof(int)), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipMalloc((void **)&dev_c, N * sizeof(int)), CCUDAEventGroupImpl);

        CudaSafeCallEx(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAEventGroupImpl);
        if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
        {
            throw CCUDAEventGroupImpl("hipHostAlloc", "CPU 内存开辟失败!");
        }
        for (int i = 0; i < FULL_DATA_SIZE; i++)
        {
            host_a[i] = i;
            host_b[i] = FULL_DATA_SIZE - i;
        }

        CudaSafeCallEx(hipEventRecord(start_event), CCUDAEventGroupImpl);
        for (int i = 0; i < FULL_DATA_SIZE; i += N)
        {
            CudaSafeCallEx(hipMemcpy(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice), CCUDAEventGroupImpl);
            CudaSafeCallEx(hipMemcpy(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice), CCUDAEventGroupImpl);

            kernelAdd_event<<<N / 1024, 1024>>>(dev_a, dev_b, dev_c);

            CudaSafeCallEx(hipMemcpy(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost), CCUDAEventGroupImpl);
        }
        CudaSafeCallEx(hipEventSynchronize(start_event), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipEventRecord(end_event), CCUDAEventGroupImpl);

        float time_gpu;
        CudaSafeCallEx(hipEventElapsedTime(&time_gpu, start_event, end_event), CCUDAEventGroupImpl);

        // 事件计时
        //printf("GPU time: %.3f ms \n", time_gpu);

        (CTestBase *)(this)->Log("CCUDAEventGroupImpl excute success...");

        CudaSafeCallEx(hipHostFree(host_a), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipHostFree(host_b), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipHostFree(host_c), CCUDAEventGroupImpl);

        CudaSafeCallEx(hipFree(dev_a), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipFree(dev_b), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipFree(dev_c), CCUDAEventGroupImpl);

        CudaSafeCallEx(hipEventDestroy(start_event), CCUDAEventGroupImpl);
        CudaSafeCallEx(hipEventDestroy(end_event), CCUDAEventGroupImpl);
        (CTestBase *)(this)->Log("CCUDAEventGroupImpl out...");
        //this->PrintLog();
        END_EXCEPTION(CTestBase);
        CudaKernelCheck();
        return true;
    }

    bool CCUDAEventGroupImpl::SetupBenchMark()
    {
        return true;
    }
}
