#include "hip/hip_runtime.h"

#include "cudaInc.h"
#include <iostream>
#include "CCUDAEventGroupImpl.h"
#include "CTestCUDAFactory.h"
#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20
namespace TestCUDA
{
    REGISTER_TEST_OBJECT(CCUDAEventGroupImpl, CUDA_EVENT_GROUP);
    __global__ void kernelAdd_event(int *a, int *b, int *c)
    {
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < N)
        {
            c[threadID] = (a[threadID] + b[threadID]) / 2;
        }
    }
    void CCUDAEventGroupImpl::Test_normal()
    {
#ifdef _DEBUG
        LOG_DEBUG("CCUDAEventGroupImpl in...");
#endif
        hipEvent_t start_event, end_event;
        CudaIsSafeCall(hipEventCreate(&start_event), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipEventCreate(&end_event), CCUDAEventGroupImpl);
#ifdef _DEBUG
        LOG_DEBUG("CCUDAEventGroupImpl create success...");
#endif
        int *host_a, *host_b, *host_c;
        int *dev_a, *dev_b, *dev_c;

        CudaIsSafeCall(hipMalloc((void **)&dev_a, N * sizeof(int)), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipMalloc((void **)&dev_b, N * sizeof(int)), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipMalloc((void **)&dev_c, N * sizeof(int)), CCUDAEventGroupImpl);

        CudaIsSafeCall(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAEventGroupImpl);
        if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
        {
            throw CCUDAEventGroupImpl("hipHostAlloc", "CPU 内存开辟失败!");
        }
        for (int i = 0; i < FULL_DATA_SIZE; i++)
        {
            host_a[i] = i;
            host_b[i] = FULL_DATA_SIZE - i;
        }

        CudaIsSafeCall(hipEventRecord(start_event), CCUDAEventGroupImpl);
        for (int i = 0; i < FULL_DATA_SIZE; i += N)
        {
            CudaIsSafeCall(hipMemcpy(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice), CCUDAEventGroupImpl);
            CudaIsSafeCall(hipMemcpy(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice), CCUDAEventGroupImpl);

            kernelAdd_event<<<N / 1024, 1024>>>(dev_a, dev_b, dev_c);

            CudaIsSafeCall(hipMemcpy(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost), CCUDAEventGroupImpl);
        }
        CudaIsSafeCall(hipEventSynchronize(start_event), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipEventRecord(end_event), CCUDAEventGroupImpl);

        float time_gpu;
        CudaIsSafeCall(hipEventElapsedTime(&time_gpu, start_event, end_event), CCUDAEventGroupImpl);

        // 事件计时
        // printf("GPU time: %.3f ms \n", time_gpu);
#ifdef _DEBUG
        LOG_DEBUG("CCUDAEventGroupImpl excute success...");
#endif
        CudaIsSafeCall(hipHostFree(host_a), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipHostFree(host_b), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipHostFree(host_c), CCUDAEventGroupImpl);

        CudaIsSafeCall(hipFree(dev_a), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipFree(dev_b), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipFree(dev_c), CCUDAEventGroupImpl);

        CudaIsSafeCall(hipEventDestroy(start_event), CCUDAEventGroupImpl);
        CudaIsSafeCall(hipEventDestroy(end_event), CCUDAEventGroupImpl);
#ifdef _DEBUG
        LOG_DEBUG("CCUDAEventGroupImpl out...");
#endif
    }
    void CCUDAEventGroupImpl::Test_NonNormal()
    {
    }

}
