#include "hip/hip_runtime.h"
#include "cudaInc.h"
#include "CBenchMark_CUDAStreamCreateImpl.h"
#define N (1024*1024)  
#define FULL_DATA_SIZE N*20  
__global__ void kernelAdd(int* a, int *b, int*c)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (threadID < N)
	{
		c[threadID] = (a[threadID] + b[threadID]) / 2;
	}
}

CBenchMarkBase *CBenchMark_CUDAStreamCreateImpl::CreateSelf()
{
	return new CBenchMark_CUDAStreamCreateImpl();
}
void CBenchMark_CUDAStreamCreateImpl::DestoryPtr(CBenchMarkBase *_ptr){
	if(_ptr != nullptr){
		delete _ptr;
		_ptr = nullptr;
	}
}

bool CBenchMark_CUDAStreamCreateImpl::SetupTest(){
	this->Log("hipStreamCreate in...");

    hipStream_t stream;
	CudaSafeCall(hipStreamCreate(&stream));

	this->Log("hipStreamCreate create success...");

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;
 
	CudaSafeCall(hipMalloc((void**)&dev_a, N * sizeof(int)));
	CudaSafeCall(hipMalloc((void**)&dev_b, N * sizeof(int)));
	CudaSafeCall(hipMalloc((void**)&dev_c, N * sizeof(int)));

	CudaSafeCall(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
	CudaSafeCall(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
	CudaSafeCall(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
 
	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE - i;
	}
 
	for (int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		CudaSafeCall(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
		CudaSafeCall(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
 
		kernelAdd << <N / 1024, 1024, 0, stream >> > (dev_a, dev_b, dev_c);
 
		CudaSafeCall(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
	}
 
	CudaSafeCall(hipStreamSynchronize(stream));
	this->Log("hipStreamCreate excute success...");

	CudaSafeCall(hipHostFree(host_a));
	CudaSafeCall(hipHostFree(host_b));
	CudaSafeCall(hipHostFree(host_c));
 
	CudaSafeCall(hipFree(dev_a));
	CudaSafeCall(hipFree(dev_b));
	CudaSafeCall(hipFree(dev_c));
 
	CudaSafeCall(hipStreamDestroy(stream));
	this->Log("hipStreamCreate out...");
    return true;
}

bool CBenchMark_CUDAStreamCreateImpl::SetupBenchMark(){


	return true;
}
bool CBenchMark_CUDAStreamCreateImpl::SetName(const char *_api_name){
	this->m_api_name = std::string(_api_name);
	return true;
}