#include "hip/hip_runtime.h"
#include "cudaInc.h"
#include <iostream>
#include "CCUDAStreamGroupImpl.h"
#include "CTestCUDAFactory.h"
#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20
namespace TestCUDA
{
	REGISTER_TEST_OBJECT(CCUDAStreamGroupImpl, CUDA_STREAM_GROUP);
	__global__ void kernelAdd(int *a, int *b, int *c)
	{
		int threadID = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadID < N)
		{
			c[threadID] = (a[threadID] + b[threadID]) / 2;
		}
	}
	void CCUDAStreamGroupImpl::Test_normal()
	{
		(CTestBase *)(this)->Log("hipStreamCreate in...");

		hipStream_t stream;
		CudaSafeCallEx(hipStreamCreate(&stream), CCUDAStreamGroupImpl);

		(CTestBase *)(this)->Log("hipStreamCreate create success...");

		int *host_a, *host_b, *host_c;
		int *dev_a, *dev_b, *dev_c;

		CudaSafeCallEx(hipMalloc((void **)&dev_a, N * sizeof(int)), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipMalloc((void **)&dev_b, N * sizeof(int)), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipMalloc((void **)&dev_c, N * sizeof(int)), CCUDAStreamGroupImpl);

		CudaSafeCallEx(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAStreamGroupImpl);
		if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
		{
			throw CCUDAStreamGroupImpl("hipHostAlloc", "CPU 内存开辟失败!");
		}
		for (int i = 0; i < FULL_DATA_SIZE; i++)
		{
			host_a[i] = i;
			host_b[i] = FULL_DATA_SIZE - i;
		}

		for (int i = 0; i < FULL_DATA_SIZE; i += N)
		{
			CudaSafeCallEx(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream), CCUDAStreamGroupImpl);
			CudaSafeCallEx(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream), CCUDAStreamGroupImpl);

			kernelAdd<<<N / 1024, 1024, 0, stream>>>(dev_a, dev_b, dev_c);

			CudaSafeCallEx(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream), CCUDAStreamGroupImpl);
		}

		CudaSafeCallEx(hipStreamSynchronize(stream), CCUDAStreamGroupImpl);
		(CTestBase *)(this)->Log("hipStreamCreate excute success...");

		CudaSafeCallEx(hipHostFree(host_a), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipHostFree(host_b), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipHostFree(host_c), CCUDAStreamGroupImpl);

		CudaSafeCallEx(hipFree(dev_a), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipFree(dev_b), CCUDAStreamGroupImpl);
		CudaSafeCallEx(hipFree(dev_c), CCUDAStreamGroupImpl);

		CudaSafeCallEx(hipStreamDestroy(stream), CCUDAStreamGroupImpl);
		(CTestBase *)(this)->Log("hipStreamCreate out...");
		// this->PrintLog();
		CudaKernelCheck();
	}
	void CCUDAStreamGroupImpl::Test_NonNormal(){

	}
	bool CCUDAStreamGroupImpl::SetupTest()
	{
		BEGIN_EXCEPTION
		Test_normal();
		Test_NonNormal();
		END_EXCEPTION(CTestBase);
		return true;
	}

	bool CCUDAStreamGroupImpl::SetupBenchMark()
	{

		return true;
	}

}
