#include "hip/hip_runtime.h"
#include "cudaInc.h"
#include <iostream>
#include "CCUDAStreamGroupImpl.h"
#include "CTestCUDAFactory.h"
#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20
namespace TestCUDA
{
	REGISTER_TEST_OBJECT(CCUDAStreamGroupImpl, CUDA_STREAM_GROUP);
	__global__ void kernelAdd(int *a, int *b, int *c)
	{
		int threadID = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadID < N)
		{
			c[threadID] = (a[threadID] + b[threadID]) / 2;
		}
	}
	void CCUDAStreamGroupImpl::Test_normal()
	{
#ifdef _DEBUG
		LOG_DEBUG("hipStreamCreate in...");
#endif
		hipStream_t stream;
		CudaIsSafeCall(hipStreamCreate(&stream), CCUDAStreamGroupImpl);
#ifdef _DEBUG
		LOG_DEBUG("hipStreamCreate create success...");
#endif
		int *host_a, *host_b, *host_c;
		int *dev_a, *dev_b, *dev_c;

		CudaIsSafeCall(hipMalloc((void **)&dev_a, N * sizeof(int)), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipMalloc((void **)&dev_b, N * sizeof(int)), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipMalloc((void **)&dev_c, N * sizeof(int)), CCUDAStreamGroupImpl);

		CudaIsSafeCall(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault), CCUDAStreamGroupImpl);
		if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
		{
			throw CCUDAStreamGroupImpl("hipHostAlloc", "CPU 内存开辟失败!");
		}
		for (int i = 0; i < FULL_DATA_SIZE; i++)
		{
			host_a[i] = i;
			host_b[i] = FULL_DATA_SIZE - i;
		}
		hipEvent_t start_event;
		CudaIsSafeCall(hipEventCreate(&start_event), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipEventRecord(start_event), CCUDAStreamGroupImpl);

		for (int i = 0; i < FULL_DATA_SIZE; i += N)
		{
			CudaIsSafeCall(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream), CCUDAStreamGroupImpl);
			CudaIsSafeCall(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream), CCUDAStreamGroupImpl);

			kernelAdd<<<N / 1024, 1024, 0, stream>>>(dev_a, dev_b, dev_c);

			CudaIsSafeCall(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream), CCUDAStreamGroupImpl);
		}
		CudaIsSafeCall(hipEventSynchronize(start_event), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipStreamSynchronize(stream), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipStreamWaitEvent(stream, start_event), CCUDAStreamGroupImpl);
#ifdef _DEBUG
		LOG_DEBUG("hipStreamCreate excute success...");
#endif
		CudaIsSafeCall(hipHostFree(host_a), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipHostFree(host_b), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipHostFree(host_c), CCUDAStreamGroupImpl);

		CudaIsSafeCall(hipFree(dev_a), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipFree(dev_b), CCUDAStreamGroupImpl);
		CudaIsSafeCall(hipFree(dev_c), CCUDAStreamGroupImpl);

		CudaIsSafeCall(hipStreamDestroy(stream), CCUDAStreamGroupImpl);
#ifdef _DEBUG
		LOG_DEBUG("hipStreamCreate out...");
#endif
		CudaKernelCheck();
	}
	void CCUDAStreamGroupImpl::Test_NonNormal()
	{
	}

}
